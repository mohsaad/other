#include "hip/hip_runtime.h"
#include "interlacing.h"

/*
    TODO:
    * write test cases for differing heights/widths
    * write test cases for null pointers
    * write test case for kernels

*/

__global__ void flip_image_kernel(uint8_t* input_image, size_t num_rows, size_t num_cols, size_t num_elements)
{
    uint8_t temp;
    int ty = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = blockIdx.y * blockDim.y + threadIdx.y;

    size_t pixel_ind = ty*num_rows + tx;

    if(pixel_ind < num_elements /  2)
    {
        temp = input_image[pixel_ind];
        input_image[pixel_ind] = input_image[num_elements - (pixel_ind + 1)];
        input_image[num_elements - (pixel_ind + 1)] = temp;
    }
}

Interlacer::Interlacer()
{
    // initialize everything to just a basic constructor, no arguments, everything is null
    reset_videos();
    height = 0;
    width = 0;
    video_1 = NULL;
    video_2 = NULL;
    output_video = NULL;
}

Interlacer::~Interlacer()
{
    // call reset_videos to do the dirty work
    reset_videos();
}

Interlacer::Interlacer(const Interlacer & interlace_obj)
{
    reset_videos();
    video_1 = interlace_obj.video_1;
    video_2 = interlace_obj.video_2;
    height = interlace_obj.height;
    width = interlace_obj.width;
}

Interlacer::Interlacer(const string & video_1_name, const string & video_2_name, const string & video_output_name)
{
    reset_videos();
    initialize_interlacer(video_1_name, video_2_name, video_output_name);
}

void Interlacer::initialize_interlacer(const string & video_1_name, const string & video_2_name, const string & video_output_name)
{
    reset_videos();

    read_first_video(video_1_name);
    read_second_video(video_2_name);

    // set appropriate height and width
    height = std::max(video_1->get(CV_CAP_PROP_FRAME_HEIGHT), video_2->get(CV_CAP_PROP_FRAME_HEIGHT));
    width = std::max(video_1->get(CV_CAP_PROP_FRAME_WIDTH), video_2->get(CV_CAP_PROP_FRAME_WIDTH));

    initialize_output_video(video_output_name);

}

void Interlacer::reset_videos()
{
    if(video_1 != NULL)
    {
        video_1 -> release();
        delete video_1;
        video_1 = NULL;
    }
    if(video_2 != NULL)
    {
        video_2 -> release();
        delete video_2;
        video_2 = NULL;
    }
    if(output_video != NULL)
    {
        output_video->release();
        delete output_video;
        output_video = NULL;
    }
}

void Interlacer::interlace()
{
    Mat video_frame_1, video_frame_2;
    Mat gray_frame_1, gray_frame_2;
    Mat resized_frame_1, resized_frame_2;

    // create a stream
    hipStreamCreate(&stream);
    //
    uint8_t * image_1;
    gpuErrchk(hipMalloc((void**)&image_1, sizeof(uint8_t)*height*width));

    // grid and block dimensions for the video
    dim3 dimGrid(ceil(height * 1.0/BLOCK_SIZE), ceil(width * 1.0/BLOCK_SIZE), 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    int i = 0;
    // loop through each frame until we have
    while(video_1->read(video_frame_1) && video_2->read(video_frame_2))
    {
        // convert to grayscale in case it already isn't
        cvtColor(video_frame_1, gray_frame_1, CV_BGR2GRAY);
        cvtColor(video_frame_2, gray_frame_2, CV_BGR2GRAY);

        // resize to max resolution
        resize(gray_frame_1, resized_frame_1, cv::Size(width, height));
        resize(gray_frame_2, resized_frame_2, cv::Size(width, height));

        // copy first image to GPU
        hipMemcpyAsync(image_1, (uint8_t*)resized_frame_1.data, sizeof(uint8_t)*height*width, hipMemcpyHostToDevice, stream);
        // execute kernel, doesn't block so we can continue doing work on CPU
        flip_image_kernel<<<dimGrid, dimBlock, 0, stream>>>(image_1, width, height, height*width);

        // flip the second image
        flip_image(resized_frame_2);

        // copy the kernel code which should be done by now - in case it isn't, we block until previous kernel calls finish
        hipMemcpy((uint8_t*)resized_frame_1.data, image_1, sizeof(uint8_t)*height*width, hipMemcpyDeviceToHost);
        output_video->write(resized_frame_1);
        output_video->write(resized_frame_2);

        // synchronize devices
        hipDeviceSynchronize();
        i++;
        std::cout << "Processed frames " << i << std::endl;
    }

    hipFree(image_1);
    hipStreamDestroy(stream);

}

void Interlacer::flip_image(Mat & image)
{
    // get some parameters relating to the data,
    // including the number of channels and total
    // elements in the data
    int num_channels = image.channels();
    uint8_t* imdata = (uint8_t*)image.data;
    size_t num_elements = image.total();

    // We're going to do this by linearizing the entire data
    // and flipping the first and last elements, and then iterating by
    // num_channels. This allows us to still remain in O(n) time,
    // but also handle color or grayscale images regardless of
    // the image type
    // i points to a block of elements in data, and j points to each
    // element in that block. We then swap the i+jth element with
    // it's corresponding element at the end of the array, addressed by
    // (num_elements - (num_channels)*(i+1) + j
    for(size_t i = 0; i < (num_elements / num_channels) / 2; i += num_channels)
    {
        for(int j = 0; j < num_channels; j++)
        {
            std::swap(imdata[i+j], imdata[num_elements - num_channels*(i+1) + j]);
        }
    }

}

void Interlacer::read_first_video(const string & video_1_name)
{
    video_1 = new VideoCapture();
    if(!video_1->open(video_1_name))
    {
        std::cout << "No file found for " << video_1_name << "!" << std::endl;
        reset_videos();
        exit(-1);
    }

}

void Interlacer::read_second_video(const string & video_2_name)
{
    video_2 = new VideoCapture();
    if(!video_2->open(video_2_name))
    {
        std::cout << "No file found for " << video_2_name << "!" << std::endl;
        reset_videos();
        exit(-1);
    }
}

void Interlacer::initialize_output_video(const string & video_output_name)
{
    output_video = new VideoWriter();
    if(!output_video->open(video_output_name, CV_FOURCC('M','J','P','G'), 10, cv::Size(width, height), false))
    {
        std::cout << "Bad file initialization!" << std::endl;
        reset_videos();
        exit(-1);
    }
}
